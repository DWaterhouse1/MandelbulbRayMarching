#include "hip/hip_runtime.h"
#include "RayMarchCompute.cuh"

#define USE_GPU_TRIG

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "Shading.cuh"

// nvcc will compile qualified namespaces, but it breaks intellisense
namespace rmcuda
{
namespace compute
{
//const float3 dummyColour = make_float3(0.0f);
static constexpr float3 dummyColour = { 0.0f };

__device__ float3 march(Ray ray, float exponent, float3 inColour);
__device__ float sphereDistance(float3 position, float3 centre, float radius);
__device__ float3 sphereNormal(float3 pos, float3 center, float radius);
__device__ float mandelbulbDistance(float3 position, float exponent);
__device__ float3 mandelbulbNormal(float3 pos, float exponent);

// TODO respect differing aspect ratios
template <typename ShadingPolicy>
__global__ void rayMarch(
	hipSurfaceObject_t surface,
	dim3 pixelDim,
	Camera camera,
	float exponent,
	int numSamples,
	float3 inColourA,
	float3 inColourB)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= pixelDim.x || y >= pixelDim.y) return;

	//TODO seed this with a random value
	hiprandState state;
	hiprand_init(1729, 0, 0, &state);

	float3 color = make_float3(0.0f);

	for (int i = 0; i < numSamples; ++i)
	{
		// mapping (0, width) X (0, height) -> (-1, 1)^2
		float xNorm = (float(x + hiprand_uniform(&state)) / float(pixelDim.x) - 0.5f) * 2.0f;
		float yNorm = (float(y + hiprand_uniform(&state)) / float(pixelDim.y) - 0.5f) * 2.0f;

		Ray ray =
		{
			camera.pos,
			normalize(
				camera.right * xNorm +
				camera.up * yNorm +
				camera.dir * camera.invhalffov)
		};

		color += testMarch<ShadingPolicy>(ray, exponent, inColourA, inColourB);
	}

	color /= numSamples;

	if (x < pixelDim.x && y < pixelDim.y)
	{
		uchar4 data = make_uchar4(
			color.x * 255,
			color.y * 255,
			color.z * 255,
			255);
		surf2Dwrite(data, surface, x * sizeof(uchar4), y);
	}
}

void basicRayMarching(hipSurfaceObject_t surface, dim3 texDim, Camera camera, float exponent, int numSamples)
{
	dim3 thread(16, 16);
	dim3 block(texDim.x / thread.x, texDim.y / thread.y);
	rayMarch<Diffuse><<<block, thread>>>(
		surface,
		texDim,
		camera,
		exponent,
		numSamples,
		dummyColour,
		dummyColour);
}

void rayMarchDiffuseColour(
	hipSurfaceObject_t surface,
	dim3 texDim,
	Camera camera,
	float exponent,
	int numSamples,
	float3 colour)
{
	dim3 thread(16, 16);
	dim3 block(texDim.x / thread.x, texDim.y / thread.y);
	rayMarch<Diffuse><<<block, thread>>>(
		surface,
		texDim,
		camera,
		exponent,
		numSamples,
		colour,
		dummyColour);
}

void rayMarchNormalColour(
	hipSurfaceObject_t surface,
	dim3 texDim,
	Camera camera,
	float exponent,
	int numSamples)
{
	dim3 thread(16, 16);
	dim3 block(texDim.x / thread.x, texDim.y / thread.y);
	rayMarch<Normal><<<block, thread>>>(
		surface,
		texDim,
		camera,
		exponent,
		numSamples,
		dummyColour,
		dummyColour);
}

void rayMarchStepwiseColour(
	hipSurfaceObject_t surface,
	dim3 texDim,
	Camera camera,
	float exponent,
	int numSamples,
	float3 lowColour,
	float3 highColour)
{
	dim3 thread(16, 16);
	dim3 block(texDim.x / thread.x, texDim.y / thread.y);
	rayMarch<Stepwise><<<block, thread>>>(
		surface,
		texDim,
		camera,
		exponent,
		numSamples,
		lowColour,
		highColour);
}

__device__ float3 march(Ray ray, float exponent, float3 inColour)
{
	float totalDistance = 0.0f;
	const float minDistance = 0.0001f;
	const float maxDistance = 100.0f;
	const int maxSteps = 2000;

	const float3 sphereCenter = make_float3(0.0f);

	for (int i = 0; i < maxSteps; ++i)
	{
		float3 currentPosition = ray.origin + (totalDistance * ray.direction);

		//float stepDistance = sphereDistance(currentPosition, sphereCenter, 1.0f);
		float stepDistance = mandelbulbDistance(currentPosition, exponent);

		if (stepDistance < minDistance)
		{
			// hit
#if 1
			float3 normal = mandelbulbNormal(currentPosition, exponent);

			float3 lightPosition = make_float3(2.0, -5.0, 3.0);

			float3 lightDirection = normalize(currentPosition - lightPosition);

			float intensity = max(0.0f, dot(normal, lightDirection));

			return inColour * intensity;
#else
			return 0.5f * mandelbulbNormal(currentPosition, exponent) + 0.5f;
#endif
		}

		if (totalDistance > maxDistance) break;

		totalDistance += stepDistance;
	}

	// didn't hit
	return make_float3(0.05f, 0.05f, 0.05f);
}

__device__ float mandelbulbDistance(float3 position, float exponent)
{
	float3 z = position;

	const int maxItt = 20;
	//const float exponent = 9.0f;

	//int steps = 0;

	float dr = 1.0f;
	float r = 0.0f;

	for (int i = 0; i < maxItt; ++i)
	{
		//steps = i;
		r = length(z);
		if (r > 4.0f) break;

		// convert to spherical coordinates
		float theta = acos(z.z / r);
		float phi = atan(z.y / z.x);
		dr = pow(r, exponent - 1.0f) * exponent * dr + 1.0f;

		// scale and rotate
		float zr = pow(r, exponent);
		theta *= exponent;
		phi *= exponent;

		// convert back to cartesian
		z = zr * make_float3(
			sin(theta) * cos(phi),
			sin(theta) * sin(phi),
			cos(theta));

		z += position;
	}

	return 0.5f * log(r) * r / dr;
}

__device__ float3 mandelbulbNormal(float3 pos, float exponent)
{
	const float peturb = 1e4;

	float fx =
		mandelbulbDistance(make_float3(pos.x + peturb, pos.y, pos.z), exponent) -
		mandelbulbDistance(make_float3(pos.x - peturb, pos.y, pos.z), exponent);
	float fy =
		mandelbulbDistance(make_float3(pos.x, pos.y + peturb, pos.z), exponent) -
		mandelbulbDistance(make_float3(pos.x, pos.y - peturb, pos.z), exponent);
	float fz =
		mandelbulbDistance(make_float3(pos.x, pos.y, pos.z + peturb), exponent) -
		mandelbulbDistance(make_float3(pos.x, pos.y, pos.z - peturb), exponent);

	return normalize(make_float3(fx, fy, fz));
}

__device__ float3 sphereNormal(float3 pos, float3 center, float radius)
{
	const float peturb = 1e4;

	float fx =
		sphereDistance(make_float3(pos.x + peturb, pos.y, pos.z), center, radius) -
		sphereDistance(make_float3(pos.x - peturb, pos.y, pos.z), center, radius);
	float fy =
		sphereDistance(make_float3(pos.x, pos.y + peturb, pos.z), center, radius) -
		sphereDistance(make_float3(pos.x, pos.y - peturb, pos.z), center, radius);
	float fz =
		sphereDistance(make_float3(pos.x, pos.y, pos.z + peturb), center, radius) -
		sphereDistance(make_float3(pos.x, pos.y, pos.z - peturb), center, radius);

	return normalize(make_float3(fx, fy, fz));
}

__device__ float sphereDistance(float3 position, float3 center, float radius)
{
	return length(position - center) - radius;
}
} // namespace rmcuda
} // namespace compute